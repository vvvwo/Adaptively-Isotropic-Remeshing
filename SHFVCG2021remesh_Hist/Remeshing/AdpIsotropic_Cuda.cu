#include "hip/hip_runtime.h"  
  
#include <vector>

__global__ void AnIsotropic_Unit_Cuda_Block(
	float** ps_2d_Cu,
	float* pt_para_Cu,
	int* XYZNumber_Cu,
	int* pt_VoxelJudge_Cu,
	int voxelSize,
	int length,//ps_2d_Cu length for a center
	int centerNum,//source center number
	float rate,
	float* resultMatching_Cu
) {

}

extern "C" void AnIsotropic_Unit_Cuda(
	std::vector<std::vector<double>> points,//source point cloud with different centers	
	std::vector<std::vector<int>> pointNeighbor,//parameter of target point cloud voxel structure: minXYZ,maxXYZ,unitSize
	std::vector<double> cu_ave,//parameter of target point cloud voxel structure: XYZNumber
	std::vector<int> points_Keep
) {



}